#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>
#include "naive_dd_common.h"

using namespace std;

#define CUDAERR_RET(str) if (str != hipSuccess) return ERROR

int CopyFrameInfoToDevice(FrameInfo fi, FrameInfo * fi_device) {

    //Allocate device memory
    CUDAERR_RET(hipMalloc(
        reinterpret_cast<void **>(&(fi_device->stdData)), fi.stdDataSize * sizeof(dtype)));
    fi_device->stdDataSize = fi.stdDataSize;

    CUDAERR_RET(hipMalloc(
	reinterpret_cast<void **>(&(fi_device->startIndices)), fi.frameCount * sizeof(int)));
    CUDAERR_RET(hipMalloc(
	reinterpret_cast<void **>(&(fi_device->stopIndices)), fi.frameCount * sizeof(int)));
    CUDAERR_RET(hipMalloc(
	reinterpret_cast<void **>(&(fi_device->dataOffset)), fi.frameCount * sizeof(int)));
    fi_device->frameCount = fi.frameCount;

    CUDAERR_RET(hipMalloc(
        reinterpret_cast<void **>(&(fi_device->subWindows)), fi.subWindowCount * sizeof(int)));
    fi_device->subWindowCount = fi.subWindowCount;

    CUDAERR_RET(hipMalloc(
        reinterpret_cast<void **>(&(fi_device->temp)), fi.frameCount * sizeof(dtype)));
    CUDAERR_RET(hipMalloc(
        reinterpret_cast<void **>(&(fi_device->temp1)), fi.frameCount * sizeof(int)));
    CUDAERR_RET(hipMalloc(
        reinterpret_cast<void **>(&(fi_device->temp2)), fi.frameCount * sizeof(int)));

    //Memory content transfer to device
    CUDAERR_RET(hipMemcpy(
        fi_device->stdData, fi.stdData, 
	fi.stdDataSize * sizeof(dtype), hipMemcpyHostToDevice));

    CUDAERR_RET(hipMemcpy(
        fi_device->startIndices, fi.startIndices,
	fi.frameCount * sizeof(int), hipMemcpyHostToDevice));
    CUDAERR_RET(hipMemcpy(
        fi_device->stopIndices, fi.stopIndices,
	fi.frameCount * sizeof(int), hipMemcpyHostToDevice));
    CUDAERR_RET(hipMemcpy(
        fi_device->dataOffset, fi.dataOffset,
	fi.frameCount * sizeof(int), hipMemcpyHostToDevice));

    CUDAERR_RET(hipMemcpy(
	fi_device->subWindows, fi.subWindows,
	fi.subWindowCount * sizeof(int), hipMemcpyHostToDevice));

    return SUCCESS;
}

__global__ void EuclideanDistance(FrameInfo fi, int knowledgeWindowSize, int subWindowIdx) {

    extern __shared__ dtype frameData[];

    int frameId     = (blockIdx.y * gridDim.x) + blockIdx.x;
    int locThreadId = (threadIdx.y * blockDim.x) + threadIdx.x;
    int tpb         = blockDim.y * blockDim.x;
    int stride      = knowledgeWindowSize / tpb;
    stride += ((knowledgeWindowSize % tpb) != 0) ? 1 : 0;

    int     frameLen   = fi.stopIndices[frameId] - fi.startIndices[frameId];
    dtype * frameStart = &fi.stdData[fi.dataOffset[frameId]];

    //Results
    dtype * atomicDist     = &(frameData[frameLen]);
    int   * atomicLock     = reinterpret_cast<int *>(&(frameData[frameLen + 1]));
    int   * atomicStartIdx = reinterpret_cast<int *>(&(frameData[frameLen + 2]));
    int   * atomicEndIdx   = reinterpret_cast<int *>(&(frameData[frameLen + 3]));

    int subWindowSize    = fi.subWindows[subWindowIdx];
    int lastSubWindowIdx = frameLen - subWindowSize;

    //Load data into shared memory
    for (int iStrd=0;iStrd<stride;iStrd++) {
        int threadId = (iStrd * tpb) + locThreadId;

        if (threadId > knowledgeWindowSize)
            continue;

        if (threadId == 0) {
            *atomicDist = 0.0;
            *atomicLock = 0;
        }
        frameData[threadId] = frameStart[threadId];
    }
    __syncthreads();

    //Start looking for anomalies
    for (int iStrd=0;iStrd<stride;iStrd++) {
        int threadId = (iStrd * tpb) + locThreadId;
        int curSubWindowIdx = threadId;
        if (threadId > lastSubWindowIdx)
            return;

        dtype curNearestDist = 999999.99e+99;
        for (int compSubWindowIdx=0;compSubWindowIdx<lastSubWindowIdx;compSubWindowIdx++) {
            if (abs(curSubWindowIdx - compSubWindowIdx) < subWindowSize)
	        continue;

            dtype sumSqErr = 0.0;
            for (int idx=0;idx<subWindowSize;idx++) {
                sumSqErr += powf(frameStart[curSubWindowIdx + idx] - 
                    frameStart[compSubWindowIdx + idx], 2.0);
            }
            dtype euclideanDist = sqrtf(sumSqErr);
            if (euclideanDist < curNearestDist)
                curNearestDist = euclideanDist;
        }
        atomicCAS(atomicLock, 0, 1);
        if (curNearestDist > *atomicDist) {
            *atomicDist     = curNearestDist;
            *atomicStartIdx = threadId;
            *atomicEndIdx   = threadId + subWindowSize;
        }
        atomicCAS(atomicLock, 1, 0);

    }
    __syncthreads();

    if (locThreadId == 0) {
        fi.temp[frameId] = *atomicDist;
        fi.temp1[frameId] = fi.startIndices[frameId] + *atomicStartIdx;
        fi.temp2[frameId] = fi.startIndices[frameId] + *atomicEndIdx;
    }
}

void HS_NaiveDetect_gpu(
    dtype * data, int dataLength,
    int ** retIndice, int * retNum,
    int ** retIndice1, int * retNum1,
    int knowledgeWindowSize, int windowStep, int minWindowSize) {

    FrameInfo fi, fi_device;

    if (CreateFrameInfo(&fi, data, dataLength,
        knowledgeWindowSize, minWindowSize, windowStep) != SUCCESS) {
        cout << "Error:      Failed to initialize" << endl;
	cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << endl;
        exit(-1);
    }
    if (CopyFrameInfoToDevice(fi, &fi_device) != SUCCESS) {
        cout << "Error:      Failed to allocate device memory" << endl;
	cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << endl;
	exit(-1);
    }

    AnomalyMatrix am;
    CreateAnomalyMatrix(fi, &am);


    hipError_t ce;
    for (int i=0;i<fi.subWindowCount;i++) {
        //Launch with: Blocks=Frames, Threads=Sub-windows
        //TODO: There is something wrong with threads=256
        dim3 blocks(fi.frameCount, 1, 1);
        EuclideanDistance<<<blocks, 1024, 
            (knowledgeWindowSize + 4) * sizeof(dtype)>>>(fi_device, knowledgeWindowSize, i);
        hipDeviceSynchronize();
        ce = hipGetLastError();

        ce = hipMemcpy(fi.temp, fi_device.temp,
            fi.frameCount * sizeof(dtype), hipMemcpyDeviceToHost);
        if (ce != hipSuccess)
            cout << "Err: " << hipGetErrorString(ce) << endl;
        ce = hipMemcpy(am.startIndices[i], fi_device.temp1, 
            fi.frameCount * sizeof(int), hipMemcpyDeviceToHost);
        if (ce != hipSuccess)
            cout << "Err: " << hipGetErrorString(ce) << endl;
        ce = hipMemcpy(am.stopIndices[i], fi_device.temp2, 
            fi.frameCount * sizeof(int), hipMemcpyDeviceToHost);
        if (ce != hipSuccess)
            cout << "Err: " << hipGetErrorString(ce) << endl;
    }

    int alarmCount = 0;
    for (int i=0;i<fi.frameCount;i++) {

        int left_bd = am.startIndices[fi.subWindowCount-1][i];
        int right_bd = am.stopIndices[fi.subWindowCount-1][i];
        int left_largest = am.startIndices[0][i];
        int right_largest = am.stopIndices[0][i];

        bool alarm = true;
        for (int j=fi.subWindowCount-2;j>=0;j--) {
            int left_test = am.startIndices[j][i];
            int right_test = am.startIndices[j][i];
            if ((left_bd <= right_test) && (right_bd >= left_test)) {
                if (left_bd < left_test) left_bd = left_test;
                if (right_bd > right_test) right_bd = right_test;
            }
            else
                alarm = false;
        }

        if (alarm) {
            fi.temp1[alarmCount] = left_largest;
            fi.temp2[alarmCount] = right_largest;
            alarmCount++;
        }
        else {
            cout << "Frame [" << i << "] dropped" << endl;
        }
    }

    cout << "    knowledge:     " << knowledgeWindowSize << endl;
    cout << "    windowStep:    " << windowStep << endl;
    cout << "    minWindowSize: " << minWindowSize << endl;
    cout << "    Execution:     " << hipGetErrorString(ce) << endl;

    *retIndice  = fi.temp1;
    *retNum     = alarmCount;
    *retIndice1 = fi.temp2;
    *retNum1    = alarmCount;

#if 1
    for (int i=0;i<alarmCount;i++)
       cout << "[eud(" << i << ")]: " << fi.temp[i] << " : " 
           << fi.startIndices[i] << " : "
	   << fi.temp1[i] << " - " << fi.temp2[i] << endl;
#endif
}

