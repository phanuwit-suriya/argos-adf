#include "hip/hip_runtime.h"
#include "dd_common.h"
#include <iostream>

using namespace std;

#ifdef DOUBLE
#define POWER pow
#else
#define POWER powf
#endif /* DOUBLE */

__device__ pdt Device_ST_Mean(pdt * data, int count) {

    pdt sum = 0.0;
    for (int idx=0;idx<count;idx++)
        sum += data[idx];

    return sum / static_cast<pdt>(count);
}

__device__ pdt Device_ST_SD(pdt * data, pdt mean, int count) {

    pdt sum = 0.0;
    for (int idx=0;idx<count;idx++)
        sum += POWER(abs(data[idx] -  mean), 2.0);

    return sqrt(sum / static_cast<pdt>(count));
}

#define DEVICE_ST(exp)      \
do {                        \
    if (threadIdx.x == 0) { \
        exp;                \
    }                       \
    __syncthreads();        \
} while (0)


__global__ void dd_DeviceProcess(
    MetricInfo * mi, AnomalyList * al, pdt * dscores) {

    __shared__ pdt iswMean;
    __shared__ pdt iswSD;
    __shared__ pdt nearestDist;
    __shared__ int lock;
#if 0
#else
    __shared__ pdt dist_s[4096];

    if (threadIdx.x == 0) {
        for (int i=0;i<mi->swPerFrameCount;i++)
            dist_s[i] = 999999999.99e+99;
    }
    __syncthreads();
#endif

    /* Frame-related info. */
    int blockId     = (blockIdx.y * gridDim.x) + blockIdx.x;
    int fIdx        = blockId / mi->swPerFrameCount;
    int fStartIndex = mi->fStartIndices[fIdx];
    //int fStopIndex  = mi->fStopIndices[fIdx];
    int fswIndex    = fIdx * mi->swPerFrameCount;
    if (blockId >= mi->fCount * mi->swPerFrameCount)
        return;

    /* Interested subwindow-related (isw) info. */
    int   iswIdx   = blockIdx.x % mi->swPerFrameCount;
    int   iswIndex = fswIndex + iswIdx;
    int   iswStartIndex = fStartIndex + mi->swStartIndices[iswIndex];
    //int   iswStopIndex  = fStartIndex + mi->swStopIndices[iswIndex];
    pdt * isw         = &(mi->data[iswStartIndex]);

    DEVICE_ST(
        iswMean = Device_ST_Mean(
            &(mi->data[iswStartIndex]), mi->options.minSubwindowSize));

    DEVICE_ST(
        iswSD = Device_ST_SD(
            &(mi->data[iswStartIndex]), iswMean, 
            mi->options.minSubwindowSize));

    DEVICE_ST(nearestDist = 999999999.99e+99);

    DEVICE_ST(lock = 0);

    __syncthreads();

    /* Comparing subwindow-related (csw) info. */
    int strides   = mi->swPerFrameCount / blockDim.x;
    strides += (mi->swPerFrameCount % blockDim.x) ? 1 : 0;

    for (int strideIdx=0;strideIdx<strides;strideIdx++) {
        int cswIdx = (strideIdx * blockDim.x) + threadIdx.x;
        if (cswIdx >= mi->swPerFrameCount)
            break;

        int cswIndex = fswIndex + cswIdx;
        int cswStartIndex = fStartIndex + mi->swStartIndices[cswIndex];
        //int cswStopIndex  = fStartIndex + mi->swStopIndices[cswIndex];

        if (abs(iswIndex - cswIndex) < mi->options.minSubwindowSize)
            continue;

        pdt cswMean = Device_ST_Mean(
            &(mi->data[cswStartIndex]), mi->options.minSubwindowSize);

        pdt cswSD = Device_ST_SD(
            &(mi->data[cswStartIndex]), 
            cswMean, mi->options.minSubwindowSize);

        pdt * csw      = &(mi->data[cswStartIndex]);
        pdt   sumSqErr = 0.0;
        for (int dIdx=0;dIdx<mi->options.minSubwindowSize;dIdx++) {
            pdt iswd = (isw[dIdx] - iswMean) / iswSD;
            pdt cswd = (csw[dIdx] - cswMean) / cswSD;
            sumSqErr += POWER(iswd - cswd, 2.0);
        }
#if 0
        pdt dist = sqrt(sumSqErr);
        while(atomicCAS(&lock, 0, 1) != 1);
        if (dist < nearestDist) {
            nearestDist = dist;
        }
        lock = 0;
#else
        dist_s[cswIdx] = sqrt(sumSqErr);
#endif
    }

    __syncthreads();

#if 0
    DEVICE_ST(
        dscores[iswIndex] = (pdt) nearestDist);
#else
    pdt nd = 999999999.99e+99;
    if (threadIdx.x == 0) {
        for (int i=0;i<mi->swPerFrameCount;i++) {
            if (dist_s[i] < nd)
                nd = dist_s[i];
        }
    }
    DEVICE_ST(
        dscores[iswIndex] = (pdt) nd);
#endif 


}

#define CHECK_ERR(exp)   \
do {                     \
    int ret = exp;       \
    if (ret) return ret; \
} while (0)

int CreateDeviceMetricInfo(MetricInfo * mi_h, MetricInfo ** mi_d) {

    MetricInfo * mi_dr, mi_hd;
    
    mi_hd = *mi_h;

    CHECK_ERR(hipMalloc(
        (void **) &mi_dr, 
        sizeof(MetricInfo)));

    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.data), 
        sizeof(pdt) * mi_h->dataLength));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.data, (void *) mi_h->data,
        sizeof(pdt) * mi_h->dataLength,
        hipMemcpyHostToDevice));
/*
    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.stdData), 
        sizeof(pdt) * mi_h->dataLength));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.stdData, (void *) mi_h->stdData,
        sizeof(pdt) * mi_h->dataLength,
        hipMemcpyHostToDevice));
*/
    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.fStartIndices), 
        sizeof(int) * mi_h->fCount));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.fStartIndices, (void *) mi_h->fStartIndices,
        sizeof(int) * mi_h->fCount,
        hipMemcpyHostToDevice));

    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.fStopIndices), 
        sizeof(int) * mi_h->fCount));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.fStopIndices, (void *) mi_h->fStopIndices,
        sizeof(int) * mi_h->fCount,
        hipMemcpyHostToDevice));

    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.swStartIndices),
        sizeof(int) * mi_h->fCount * mi_h->swPerFrameCount));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.swStartIndices, (void *) mi_h->swStartIndices,
        sizeof(int) * mi_h->fCount * mi_h->swPerFrameCount,
        hipMemcpyHostToDevice));

    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.swStopIndices),
        sizeof(int) * mi_h->fCount * mi_h->swPerFrameCount));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.swStopIndices, (void *) mi_h->swStopIndices,
        sizeof(int) * mi_h->fCount * mi_h->swPerFrameCount,
        hipMemcpyHostToDevice));

    CHECK_ERR(hipMalloc(
        (void **) &(mi_hd.swOwnerFrame),
        sizeof(int) * mi_h->fCount * mi_h->swPerFrameCount));
    CHECK_ERR(hipMemcpy(
        (void *) mi_hd.swOwnerFrame, (void *) mi_h->swOwnerFrame,
        sizeof(int) * mi_h->fCount * mi_h->swPerFrameCount,
        hipMemcpyHostToDevice));

    CHECK_ERR(hipMemcpy(
        (void *) mi_dr, (void *) &mi_hd,
        sizeof(MetricInfo),
        hipMemcpyHostToDevice));

    *mi_d = mi_dr;

    return 0;
}


int CreateHostDeviceDistanceScores(
    MetricInfo * mi, pdt ** dscores_h, pdt ** dscores_d) {

    *dscores_h = new pdt [mi->fCount * mi->swPerFrameCount];

    pdt * dscores_dr;
    CHECK_ERR(hipMalloc(
        (void **) &dscores_dr,
        sizeof(pdt) * mi->fCount * mi->swPerFrameCount));

    *dscores_d = dscores_dr;

    return 0;
}

int CreateDeviceAnomalyList(
    MetricInfo  *  mi_h,
    AnomalyList *  al_h, 
    AnomalyList ** al_d) {

    AnomalyList * al_dr, al_hd;

    CHECK_ERR(hipMalloc(
        (void **) &al_dr, sizeof(AnomalyList)));

    CHECK_ERR(hipMalloc(
        (void **) &(al_hd.startIndices),
        sizeof(int) * mi_h->fCount));

    CHECK_ERR(hipMalloc(
        (void **) &(al_hd.stopIndices),
        sizeof(int) * mi_h->fCount));

    CHECK_ERR(hipMalloc(
        (void **) &(al_hd.frameStartIndices),
        sizeof(int) * mi_h->fCount));

    CHECK_ERR(hipMalloc(
        (void **) &(al_hd.frameStopIndices),
        sizeof(int) * mi_h->fCount));

    CHECK_ERR(hipMemcpy(
        (void *) al_dr, (void *) &al_hd,
        sizeof(AnomalyList),
        hipMemcpyHostToDevice));

    *al_d = al_dr;

    return 0;
}

#undef CHECK_ERR

void dd_gpu(
    pdt *  data,                int   dataLength,
    int ** aStartIndices,       int * aStartIndicesCount,
    int ** aStopIndices,        int * aStopIndicesCount,
    int ** fStartIndices,       int * fStartIndicesCount,
    int ** fStopIndices,        int * fStopIndicesCount,
    int    knowledgeWindowSize,
    int    windowStep,
    int    minSubwindowSize,
    int    detectionFocus) {

    MetricInfo mi, * mi_d; 

    InitializeMetricDataAndOptions(
        data,
        dataLength,
        knowledgeWindowSize,
        windowStep,
        minSubwindowSize,
        detectionFocus,
        &mi);

    /* Create host and device metric list */
    //TODO: Define error code and handling here
    int ret = CreateMetricInfo(&mi);
    if (ret)
        cout << "ERROR CREATE" << endl;
    cout << "dd_gpu" << endl;
    cout << "\tknowledgeWindowSize: " << mi.options.knowledgeWindowSize << endl;
    cout << "\twindowStep:          " << mi.options.windowStep << endl;
    cout << "\tminSubwindowSize:    " << mi.options.minSubwindowSize << endl;
    cout << "\tdetectionFocus:      " << mi.options.detectionFocus << endl;

    ret = CreateDeviceMetricInfo(&mi, &mi_d);
    if (ret) {
        cout << "ERROR CREATE DEVICE METRIC INFO" << endl;
        cout << "\t" << hipGetErrorString(hipGetLastError()) << endl;
    }

    /* Create host and device anomaly list */
    AnomalyList al, * al_d;
    CreateAnomalyList(&mi, &al);
    CreateDeviceAnomalyList(&mi, &al, &al_d);

    /* Create host and device distance score arrays */
    pdt * dscores, * dscores_d;
    CreateHostDeviceDistanceScores(&mi, &dscores, &dscores_d);

    /* Launch DD kernel */
    int blockCount = mi.fCount * mi.swPerFrameCount;
    int threadCount = (mi.swPerFrameCount / 32) * 32;
    cout << "Lanching DD kernel: " << blockCount << " : "
        << threadCount << endl;

    int blockCountX = 1;
    int blockCountY = 1;
    if (blockCount > 65535) {
        blockCountY = blockCount / 65535;
        blockCountY += (blockCount % 65535) ? 1 : 0;
        blockCountX = 65535;
    }
    else {
        blockCountY = 1;
        blockCountX = blockCount;
    }
    dim3 grid(blockCountX, blockCountY, 1);
    dd_DeviceProcess<<<grid, 512>>>(mi_d, al_d, dscores_d);
    hipError_t ce = hipDeviceSynchronize();
    if (ce != hipSuccess) {
        cout << "DD Kernel Execution failed: "
            << hipGetErrorString(ce) << endl;
    }


    /* Show Distance Scores */
#if 1
    ce = hipMemcpy(dscores, dscores_d, 
        sizeof(pdt) * mi.fCount * mi.swPerFrameCount, 
        hipMemcpyDeviceToHost);
    if (ce != hipSuccess) {
        cout << "DD Transfer failed: "
            << hipGetErrorString(ce) << endl;
    }
    int count = mi.fCount * mi.swPerFrameCount;
    for (int i=0;i<count;i++)
        cout << dscores[i] << endl;
#endif

    ret = DestroyMetricInfo(&mi);
    if (ret)
        cout << "ERROR RELEASE" << endl;

    *aStartIndices = al.startIndices;
    *aStopIndices  = al.stopIndices;
    *aStartIndicesCount = 0;//mi.fCount;
    *aStopIndicesCount  = 0;//mi.fCount;
    *fStartIndices = al.frameStartIndices;
    *fStopIndices  = al.frameStopIndices;
    *fStartIndicesCount = 0;//mi.fCount;
    *fStopIndicesCount  = 0;//mi.fCount;
}

